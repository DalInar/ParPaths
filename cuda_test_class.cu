#include "hip/hip_runtime.h"
#include "cuda_test_class.h"

__global__ void add(int *a, int *b, int *c){
	c[blockIdx.x] = a[blockIdx.x] + b[blockIdx.x];
}

cuda_test_class::cuda_test_class(int N_){
	N=N_;
	a = (int *) malloc(N);
	b = (int *) malloc(N);
	c = (int *) malloc(N);

	for(int i=0; i<N; i++){
		a[i] = i;
		b[i] = i*i;
	}
}

bool cuda_test_class::check(){
	for(int i=0; i<N; i++){
		if(a[i] + b[i] != c[i]){
			std::cout<<"Error! Sum not correct for index "<<i<<std::endl;
			std::cout<<a[i]<<" + "<<b[i]<<" != "<<c[i]<<std::endl;
			return false;
		}
	}
	return true;
}

void cuda_test_class::add(){
	int *d_size;
	int *d_a, *d_b, *d_c;

	int size=N*sizeof(int);
	hipMalloc((void **) & d_a, size);
	hipMalloc((void **) & d_b, size);
	hipMalloc((void **) & d_c, size);

	//Copy inputs to device
	hipMemcpy(d_a, &a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, &b, size, hipMemcpyHostToDevice);

	add<<<N,1>>>(d_a,d_b);

	//Copy results back to host
	hipMemcpy(&c, d_c, size, hipMemcpyDeviceToHost);

	//cleanup
	hipFree(d_a); hipFree(d_b); hipFree(d_c);
}
