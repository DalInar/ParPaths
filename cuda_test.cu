

#include <hip/hip_runtime.h>
#include <iostream>

__global__ void mykernel(int *a, int * b, int * c){
	*c=*a+*b;
}

int main(void){
	int a,b,c;
	int *d_a, *d_b, *d_c;
	int size = sizeof(int);

	//Allocate space for device
	hipMalloc((void **) & d_a, size);
	hipMalloc((void **) & d_b, size);
	hipMalloc((void **) & d_c, size);

	a=2;
	b=7;

	//Copy inputs to device
	hipMemcpy(d_a, &a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, &b, size, hipMemcpyHostToDevice);

	mykernel<<<1,1>>>(d_a, d_b, d_c);

	//Copy results back to host
	hipMemcpy(&c, d_c, size, hipMemcpyDeviceToHost);

	//cleanup
	hipFree(d_a); hipFree(d_b); hipFree(d_c);

	std::cout<<"CUDA answer = "<<c<<std::endl;
	std::cout<<"Should be = "<<a+b<<std::endl;

	std::cout<<"Hello World!"<<std::endl;
	return 0;
}
