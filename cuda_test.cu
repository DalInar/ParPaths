#include "hip/hip_runtime.h"

#include <iostream>
#include "cuda_test_class.h"

__global__ void mykernel(int *a, int * b, int * c){
	*c=*a+*b;
}

int main(void){
	int a,b,c;
	int *d_a, *d_b, *d_c;
	int size = sizeof(int);

	//Allocate space for device
	hipMalloc((void **) & d_a, size);
	hipMalloc((void **) & d_b, size);
	hipMalloc((void **) & d_c, size);

	a=2;
	b=7;

	//Copy inputs to device
	hipMemcpy(d_a, &a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, &b, size, hipMemcpyHostToDevice);

	mykernel<<<1,1>>>(d_a, d_b, d_c);

	//Copy results back to host
	hipMemcpy(&c, d_c, size, hipMemcpyDeviceToHost);

	//cleanup
	hipFree(d_a); hipFree(d_b); hipFree(d_c);

	std::cout<<"CUDA answer = "<<c<<std::endl;
	std::cout<<"Should be = "<<a+b<<std::endl;

	int N=10;
	cuda_test_class c_test(N);
	std::cout<<"Created class"<<std::endl;
	std::cout<<"Now check"<<std::endl;
	std::cout<<"Passed? "<<c_test.check()<<std::endl;
	std::cout<<"GPU adding"<<std::endl;
	c_test.add();
	std::cout<<"Now check gpu"<<std::endl;
	std::cout<<"Passed? "<<c_test.check()<<std::endl;

	std::cout<<"Hello World!"<<std::endl;
	return 0;
}
