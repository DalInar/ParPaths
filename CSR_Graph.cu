#include "hip/hip_runtime.h"
/*
 * CSR_Graph.cu
 *
 *  Created on: Dec 12, 2014
 *      Author: pakij
 */

#include "CSR_Graph.h"

__global__ void test_add(int *a, int *b, int *c){
	c[blockIdx.x] = a[blockIdx.x] + b[blockIdx.x];
}

void CSR_Graph::BellmanFordGPU(int source_, std::vector <int> &predecessors, std::vector <double> &path_weight){

	int N=1000;
	int *a, *b, *c;
	int *d_a, *d_b, *d_c;

	int size=N*sizeof(int);
	a = (int *) malloc(size);
	b = (int *) malloc(size);
	c = (int *) malloc(size);
	hipMalloc((void **) & d_a, size);
	hipMalloc((void **) & d_b, size);
	hipMalloc((void **) & d_c, size);

	std::cout<<std::endl<<"GPU output"<<std::endl;
	for(int i=0; i<N; i++){
		a[i]=i;
		b[i]=i*i;
	}

	//Copy inputs to device
	hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

	test_add<<<N,1>>>(d_a,d_b,d_c);

	//Copy results back to host
	hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);


	std::cout<<std::endl<<"GPU output"<<std::endl;
	for(int i=0; i<N; i++){
		std::cout<<c[i]<<" ?= "<<a[i]+b[i]<<std::endl;
	}

	//cleanup
	hipFree(d_a); hipFree(d_b); hipFree(d_c);
	free(a); free(b); free(c);
}



