#include "hip/hip_runtime.h"
/*
 * CSR_Graph.cu
 *
 *  Created on: Dec 12, 2014
 *      Author: pakij
 */

#include "CSR_Graph.h"

__global__ void BellmanFord_split1cuda(int V, int E, int *offsets, int *edge_dests, double *weights, int * preds, int * temp_preds, double * path_weights){
	//int my_vert = blockIdx.x;
	int my_vert = blockIdx.x *blockDim.x + threadIdx.x;

	if(my_vert < V) {
		int my_dist;
		int first_target_index, last_target_index, target_index, target;
		double new_dist;
		my_dist = path_weights[my_vert];

		//Find bounds of adjacency list
		first_target_index = offsets[my_vert];
		if(my_vert != V-1){
			last_target_index = offsets[my_vert+1];
		}
		else{
			last_target_index = E;
		}

		for(target_index = first_target_index; target_index < last_target_index; target_index++){
			target = edge_dests[target_index];
			new_dist = my_dist + weights[target_index];
			// need to change path_weights[target] and update predecessors[target]
			if(new_dist < path_weights[target]){
				temp_preds[target] = my_vert;
			}
		}
	}
}

__global__ void BellmanFord_split2cuda(int V, int E, int *offsets, int *edge_dests, double *weights, int * preds, int * temp_preds, double * path_weights){
	//int my_vert = blockIdx.x;
	int my_vert = blockIdx.x *blockDim.x + threadIdx.x;
	int first_target_index, last_target_index;
	int pred_vert;

	if(my_vert < V){
		pred_vert = temp_preds[my_vert];
		if(pred_vert > 0 && pred_vert != my_vert){
			//Update predecessors
			preds[my_vert] = pred_vert;

			//Find bounds of adjacency list
			first_target_index = offsets[my_vert];
			if(my_vert != V-1){
				last_target_index = offsets[my_vert+1];
			}
			else{
				last_target_index = E;
			}

			//Update path_weights
			for(int i=first_target_index; i < last_target_index; i++){
				if(edge_dests[i] == my_vert){
					path_weights[my_vert] = path_weights[pred_vert] + weights[i];
					break;
				}
			}
		}
	}
}


double CSR_Graph::BellmanFordGPU_Split(int source_, std::vector <int> &predecessors, std::vector <double> &path_weight){
	int num_blocks = (V + threads_per_block - 1) / threads_per_block;

	//Initialize predecessor tree
	predecessors.clear();
	path_weight.clear();
	double inf = std::numeric_limits<double>::infinity();
	predecessors.resize(V,-1);
	path_weight.resize(V,E*max_weight);
	predecessors[source_]=source_;
	path_weight[source_]=0;

	boost::timer::auto_cpu_timer t;

	//GPU pointers
	int *  d_offsets;
	int * d_edge_dests;
	double * d_weights;
	int * d_predecessors;
	double * d_path_weight;
	int * d_temp_predecessors;

	//Size of CSR graph
	int offsets_size = V*sizeof(int);
	int edge_dests_size = E*sizeof(int);
	int weights_size = E*sizeof(double);

	//Size of predecessor tree into
	int predecessors_size = V*sizeof(int);
	int temp_predecessors_size = V*sizeof(int);
	int path_weight_size = V*sizeof(double);

	//Allocate memory on device
	hipMalloc((void **) & d_offsets, offsets_size);
	hipMalloc((void **) & d_edge_dests, edge_dests_size);
	hipMalloc((void **) & d_weights, weights_size);
	hipMalloc((void **) & d_predecessors, predecessors_size);
	hipMalloc((void **) & d_temp_predecessors, temp_predecessors_size);
	hipMalloc((void **) & d_path_weight, path_weight_size);

	std::cout<<"Transferring to GPU"<<std::endl;
	hipMemcpy(d_offsets, (int *) &offsets[0], offsets_size, hipMemcpyHostToDevice);
	hipMemcpy(d_edge_dests, (int *) &edge_dests[0], edge_dests_size, hipMemcpyHostToDevice);
	hipMemcpy(d_weights, (double *) &weights[0], weights_size, hipMemcpyHostToDevice);
	hipMemcpy(d_predecessors, (int *) &predecessors[0], predecessors_size, hipMemcpyHostToDevice);
	hipMemcpy(d_temp_predecessors, (int *) &predecessors[0], temp_predecessors_size, hipMemcpyHostToDevice);
	hipMemcpy(d_path_weight, (double *) &path_weight[0], path_weight_size, hipMemcpyHostToDevice);

	std::cout<<"Running kernel with <<<" << num_blocks << ", " << threads_per_block << ">>>" <<std::endl;
	boost::timer::cpu_timer timer;
	for(int iter=0; iter<V; iter++){
		//std::cout<<iter<<std::endl;
		BellmanFord_split1cuda<<<num_blocks, threads_per_block>>>(V, E, d_offsets,d_edge_dests,d_weights,d_predecessors,d_temp_predecessors,d_path_weight);
		hipDeviceSynchronize();
		BellmanFord_split2cuda<<<num_blocks, threads_per_block>>>(V, E, d_offsets,d_edge_dests,d_weights,d_predecessors,d_temp_predecessors,d_path_weight);
		hipDeviceSynchronize();
	}
	timer.stop();

	//Copy results back to host
	//hipMemcpy((int *) &offsets[0], d_offsets, offsets_size, hipMemcpyDeviceToHost);
	//hipMemcpy((int *) &edge_dests[0], d_edge_dests, edge_dests_size, hipMemcpyDeviceToHost);
	//hipMemcpy((double *) &weights[0], d_weights, weights_size, hipMemcpyDeviceToHost);
	hipMemcpy((int *) &predecessors[0], d_predecessors, predecessors_size, hipMemcpyDeviceToHost);
	hipMemcpy((double *) &path_weight[0], d_path_weight, path_weight_size, hipMemcpyDeviceToHost);

	//cleanup
	hipFree(d_offsets); hipFree(d_edge_dests); hipFree(d_weights);
	hipFree(d_predecessors); hipFree(d_path_weight); hipFree(d_temp_predecessors);

	for(int i=0; i<V; i++){
		if(path_weight[i] == E*max_weight){
			path_weight[i] = inf;
		}
	}

	return (double) timer.elapsed().wall / 1000000000.0;
}

__global__ void BellmanFord_cuda(int V, int E, int *offsets, int *edge_dests, double *weights, int * preds, double * path_weights){
	//int my_vert = blockIdx.x;
	int my_vert = blockIdx.x *blockDim.x + threadIdx.x;
	//int my_vert = threadIdx.x;

	if(my_vert < V) {
		int source_vert;

		double my_dist = path_weights[my_vert];
		double trial_dist;

		source_vert=0;
		for(int i=0; i<E; i++){
			if(edge_dests[i] == my_vert){
				//we can keep track of what the source vertex could be, since the edge list is sorted by them
				while(source_vert != V-1  && offsets[source_vert+1] <= i){
					source_vert++;
				}
				trial_dist = weights[i] + path_weights[source_vert]; //Data race, possibly benign?
				if(trial_dist < my_dist){
					path_weights[my_vert] = trial_dist;
					preds[my_vert] = source_vert;
				}
			}
		}
	}
}

double CSR_Graph::BellmanFordGPU(int source_, std::vector <int> &predecessors, std::vector <double> &path_weight){
	int num_blocks = (V + threads_per_block - 1) / threads_per_block;

	//Initialize predecessor tree
	predecessors.clear();
	path_weight.clear();
	double inf = std::numeric_limits<double>::infinity();
	predecessors.resize(V,-1);
	path_weight.resize(V,E*max_weight);
	predecessors[source_]=source_;
	path_weight[source_]=0;

	boost::timer::auto_cpu_timer t;

	//GPU pointers
	int *  d_offsets;
	int * d_edge_dests;
	double * d_weights;
	int * d_predecessors;
	double * d_path_weight;

	//Size of CSR graph
	int offsets_size = V*sizeof(int);
	int edge_dests_size = E*sizeof(int);
	int weights_size = E*sizeof(double);

	//Size of predecessor tree into
	int predecessors_size = V*sizeof(int);
	int path_weight_size = V*sizeof(double);

	//Allocate memory on device
	hipMalloc((void **) & d_offsets, offsets_size);
	hipMalloc((void **) & d_edge_dests, edge_dests_size);
	hipMalloc((void **) & d_weights, weights_size);
	hipMalloc((void **) & d_predecessors, predecessors_size);
	hipMalloc((void **) & d_path_weight, path_weight_size);

	std::cout<<"Transferring to GPU"<<std::endl;
	hipMemcpy(d_offsets, (int *) &offsets[0], offsets_size, hipMemcpyHostToDevice);
	hipMemcpy(d_edge_dests, (int *) &edge_dests[0], edge_dests_size, hipMemcpyHostToDevice);
	hipMemcpy(d_weights, (double *) &weights[0], weights_size, hipMemcpyHostToDevice);
	hipMemcpy(d_predecessors, (int *) &predecessors[0], predecessors_size, hipMemcpyHostToDevice);
	hipMemcpy(d_path_weight, (double *) &path_weight[0], path_weight_size, hipMemcpyHostToDevice);

	std::cout<<"Running kernel with <<<" << num_blocks << ", " << threads_per_block << ">>>" <<std::endl;
	boost::timer::cpu_timer timer;
	for(int iter=0; iter<V; iter++){
		//std::cout<<iter<<std::endl;
		BellmanFord_cuda<<<num_blocks, threads_per_block>>>(V, E, d_offsets,d_edge_dests,d_weights,d_predecessors,d_path_weight);
		hipDeviceSynchronize();
	}
	timer.stop();

	//Copy results back to host
	hipMemcpy((int *) &offsets[0], d_offsets, offsets_size, hipMemcpyDeviceToHost);
	hipMemcpy((int *) &edge_dests[0], d_edge_dests, edge_dests_size, hipMemcpyDeviceToHost);
	hipMemcpy((double *) &weights[0], d_weights, weights_size, hipMemcpyDeviceToHost);
	hipMemcpy((int *) &predecessors[0], d_predecessors, predecessors_size, hipMemcpyDeviceToHost);
	hipMemcpy((double *) &path_weight[0], d_path_weight, path_weight_size, hipMemcpyDeviceToHost);

	//cleanup
	hipFree(d_offsets); hipFree(d_edge_dests); hipFree(d_weights);
	hipFree(d_predecessors); hipFree(d_path_weight);

	return (double) timer.elapsed().wall / 1000000000.0;
}


//Simple test code
__global__ void test_add(int *a, int *b, int *c){
	c[blockIdx.x] = a[blockIdx.x] + b[blockIdx.x];
}

bool CSR_Graph::test_cuda(){
	int N=1000;
	int *a, *b, *c;
	int *d_a, *d_b, *d_c;
	bool result = true;

	int size=N*sizeof(int);
	a = (int *) malloc(size);
	b = (int *) malloc(size);
	c = (int *) malloc(size);
	hipMalloc((void **) & d_a, size);
	hipMalloc((void **) & d_b, size);
	hipMalloc((void **) & d_c, size);

	std::cout<<std::endl<<"GPU output"<<std::endl;
	for(int i=0; i<N; i++){
		a[i]=i;
		b[i]=i*i;
	}

	//Copy inputs to device
	hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

	test_add<<<N,1>>>(d_a,d_b,d_c);

	//Copy results back to host
	hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);


	std::cout<<std::endl<<"GPU output"<<std::endl;
	for(int i=0; i<N; i++){
		//std::cout<<c[i]<<" ?= "<<a[i]+b[i]<<std::endl;
		if(c[i] != a[i] + b[i]){
			result = false;
		}
	}

	//cleanup
	hipFree(d_a); hipFree(d_b); hipFree(d_c);
	free(a); free(b); free(c);

	return result;
}
